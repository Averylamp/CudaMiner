#include "hip/hip_runtime.h"
/*-256 tests: %s\n", sha256_test() ? "SUCCEEDED" : "FAILED");

 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */


// System includes
#include <stdio.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// helper functions and utilities to work with CUDA
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>


// TCP
#include <stdlib.h>
#include <string.h>
#include <unistd.h>
#include <sys/types.h>
#include <sys/socket.h>
#include <netinet/in.h>
#include <netdb.h>

#define BUFSIZE 1024

#ifndef MAX
#define MAX(a,b) (a > b ? a : b)
#endif


typedef unsigned int  WORD;             // 32-bit word, change to "long" for 16-bit machines

typedef struct {
	unsigned char data[64];
	WORD datalen;
	unsigned long long bitlen;
	WORD state[8];
} SHA256_CTX;

#define SHA256_BLOCK_SIZE 32            // SHA256 outputs a 32 byte digest

/*********************************************************************
* Filename:   sha256.c
* Author:     Brad Conte (brad AT bradconte.com)
* Copyright:
* Disclaimer: This code is presented "as is" without any guarantees.
* Details:    Implementation of the SHA-256 hashing algorithm.
              SHA-256 is one of the three algorithms in the SHA2
              specification. The others, SHA-384 and SHA-512, are not
              offered in this implementation.
              Algorithm specification can be found here:
               * http://csrc.nist.gov/publications/fips/fips180-2/fips180-2withchangenotice.pdf
              This implementation uses little endian byte order.
*********************************************************************/

/*************************** HEADER FILES ***************************/
#include <memory.h>

/****************************** MACROS ******************************/
#define ROTLEFT(a,b) (((a) << (b)) | ((a) >> (32-(b))))
#define ROTRIGHT(a,b) (((a) >> (b)) | ((a) << (32-(b))))

#define CH(x,y,z) (((x) & (y)) ^ (~(x) & (z)))
#define MAJ(x,y,z) (((x) & (y)) ^ ((x) & (z)) ^ ((y) & (z)))
#define EP0(x) (ROTRIGHT(x,2) ^ ROTRIGHT(x,13) ^ ROTRIGHT(x,22))
#define EP1(x) (ROTRIGHT(x,6) ^ ROTRIGHT(x,11) ^ ROTRIGHT(x,25))
#define SIG0(x) (ROTRIGHT(x,7) ^ ROTRIGHT(x,18) ^ ((x) >> 3))
#define SIG1(x) (ROTRIGHT(x,17) ^ ROTRIGHT(x,19) ^ ((x) >> 10))

/**************************** VARIABLES *****************************/
__device__ static const WORD k[64] = {
	0x428a2f98,0x71374491,0xb5c0fbcf,0xe9b5dba5,0x3956c25b,0x59f111f1,0x923f82a4,0xab1c5ed5,
	0xd807aa98,0x12835b01,0x243185be,0x550c7dc3,0x72be5d74,0x80deb1fe,0x9bdc06a7,0xc19bf174,
	0xe49b69c1,0xefbe4786,0x0fc19dc6,0x240ca1cc,0x2de92c6f,0x4a7484aa,0x5cb0a9dc,0x76f988da,
	0x983e5152,0xa831c66d,0xb00327c8,0xbf597fc7,0xc6e00bf3,0xd5a79147,0x06ca6351,0x14292967,
	0x27b70a85,0x2e1b2138,0x4d2c6dfc,0x53380d13,0x650a7354,0x766a0abb,0x81c2c92e,0x92722c85,
	0xa2bfe8a1,0xa81a664b,0xc24b8b70,0xc76c51a3,0xd192e819,0xd6990624,0xf40e3585,0x106aa070,
	0x19a4c116,0x1e376c08,0x2748774c,0x34b0bcb5,0x391c0cb3,0x4ed8aa4a,0x5b9cca4f,0x682e6ff3,
	0x748f82ee,0x78a5636f,0x84c87814,0x8cc70208,0x90befffa,0xa4506ceb,0xbef9a3f7,0xc67178f2
};

static const WORD h_k[64] = {
	0x428a2f98,0x71374491,0xb5c0fbcf,0xe9b5dba5,0x3956c25b,0x59f111f1,0x923f82a4,0xab1c5ed5,
	0xd807aa98,0x12835b01,0x243185be,0x550c7dc3,0x72be5d74,0x80deb1fe,0x9bdc06a7,0xc19bf174,
	0xe49b69c1,0xefbe4786,0x0fc19dc6,0x240ca1cc,0x2de92c6f,0x4a7484aa,0x5cb0a9dc,0x76f988da,
	0x983e5152,0xa831c66d,0xb00327c8,0xbf597fc7,0xc6e00bf3,0xd5a79147,0x06ca6351,0x14292967,
	0x27b70a85,0x2e1b2138,0x4d2c6dfc,0x53380d13,0x650a7354,0x766a0abb,0x81c2c92e,0x92722c85,
	0xa2bfe8a1,0xa81a664b,0xc24b8b70,0xc76c51a3,0xd192e819,0xd6990624,0xf40e3585,0x106aa070,
	0x19a4c116,0x1e376c08,0x2748774c,0x34b0bcb5,0x391c0cb3,0x4ed8aa4a,0x5b9cca4f,0x682e6ff3,
	0x748f82ee,0x78a5636f,0x84c87814,0x8cc70208,0x90befffa,0xa4506ceb,0xbef9a3f7,0xc67178f2
};

/*********************** FUNCTION DEFINITIONS ***********************/
 __device__ void sha256_transform(SHA256_CTX *ctx, const unsigned char data[])
{
	WORD a, b, c, d, e, f, g, h, i, j, t1, t2, m[64];

	for (i = 0, j = 0; i < 16; ++i, j += 4)
		m[i] = (data[j] << 24) | (data[j + 1] << 16) | (data[j + 2] << 8) | (data[j + 3]);
	for ( ; i < 64; ++i)
		m[i] = SIG1(m[i - 2]) + m[i - 7] + SIG0(m[i - 15]) + m[i - 16];

	a = ctx->state[0];
	b = ctx->state[1];
	c = ctx->state[2];
	d = ctx->state[3];
	e = ctx->state[4];
	f = ctx->state[5];
	g = ctx->state[6];
	h = ctx->state[7];
	for (i = 0; i < 64; ++i) {
                t1 = h + EP1(e) + CH(e,f,g) + k[i] + m[i];
		t2 = EP0(a) + MAJ(a,b,c);
		h = g;
		g = f;
		f = e;
		e = d + t1;
		d = c;
		c = b;
		b = a;
		a = t1 + t2;
	}
	ctx->state[0] += a;
	ctx->state[1] += b;
	ctx->state[2] += c;
	ctx->state[3] += d;
	ctx->state[4] += e;
	ctx->state[5] += f;
	ctx->state[6] += g;
	ctx->state[7] += h;
}

 __host__ void h_sha256_transform(SHA256_CTX *ctx, const unsigned char data[])
{
	WORD a, b, c, d, e, f, g, h, i, j, t1, t2, m[64];

	for (i = 0, j = 0; i < 16; ++i, j += 4)
		m[i] = (data[j] << 24) | (data[j + 1] << 16) | (data[j + 2] << 8) | (data[j + 3]);
	for ( ; i < 64; ++i)
		m[i] = SIG1(m[i - 2]) + m[i - 7] + SIG0(m[i - 15]) + m[i - 16];

	a = ctx->state[0];
	b = ctx->state[1];
	c = ctx->state[2];
	d = ctx->state[3];
	e = ctx->state[4];
	f = ctx->state[5];
	g = ctx->state[6];
	h = ctx->state[7];
	for (i = 0; i < 64; ++i) {
                t1 = h + EP1(e) + CH(e,f,g) + h_k[i] + m[i];
		t2 = EP0(a) + MAJ(a,b,c);
		h = g;
		g = f;
		f = e;
		e = d + t1;
		d = c;
		c = b;
		b = a;
		a = t1 + t2;
	}
	ctx->state[0] += a;
	ctx->state[1] += b;
	ctx->state[2] += c;
	ctx->state[3] += d;
	ctx->state[4] += e;
	ctx->state[5] += f;
	ctx->state[6] += g;
	ctx->state[7] += h;
}
__device__ void sha256_init(SHA256_CTX *ctx)
{
	ctx->datalen = 0;
	ctx->bitlen = 0;
	ctx->state[0] = 0x6a09e667;
	ctx->state[1] = 0xbb67ae85;
	ctx->state[2] = 0x3c6ef372;
	ctx->state[3] = 0xa54ff53a;
	ctx->state[4] = 0x510e527f;
	ctx->state[5] = 0x9b05688c;
	ctx->state[6] = 0x1f83d9ab;
	ctx->state[7] = 0x5be0cd19;
}

__device__ void sha256_update(SHA256_CTX *ctx, const char data[], size_t len)
{
	WORD i;

	for (i = 0; i < len; ++i) {
		ctx->data[ctx->datalen] = data[i];
		ctx->datalen++;
		if (ctx->datalen == 64) {
			sha256_transform(ctx, ctx->data);
			ctx->bitlen += 512;
			ctx->datalen = 0;
		}
	}
}

__device__ void sha256_final(SHA256_CTX *ctx, char hash[])
{
	WORD i;

	i = ctx->datalen;

	// Pad whatever data is left in the buffer.
	if (ctx->datalen < 56) {
		ctx->data[i++] = 0x80;
		while (i < 56)
			ctx->data[i++] = 0x00;
	}
	else {
		ctx->data[i++] = 0x80;
		while (i < 64)
			ctx->data[i++] = 0x00;
		sha256_transform(ctx, ctx->data);
		memset(ctx->data, 0, 56);
	}

	// Append to the padding the total message's length in bits and transform.
	ctx->bitlen += ctx->datalen * 8;
	ctx->data[63] = ctx->bitlen;
	ctx->data[62] = ctx->bitlen >> 8;
	ctx->data[61] = ctx->bitlen >> 16;
	ctx->data[60] = ctx->bitlen >> 24;
	ctx->data[59] = ctx->bitlen >> 32;
	ctx->data[58] = ctx->bitlen >> 40;
	ctx->data[57] = ctx->bitlen >> 48;
	ctx->data[56] = ctx->bitlen >> 56;
	sha256_transform(ctx, ctx->data);

	// Since this implementation uses little endian byte ordering and SHA uses big endian,
	// reverse all the bytes when copying the final state to the output hash.
	for (i = 0; i < 4; ++i) {
		hash[i]      = (ctx->state[0] >> (24 - i * 8)) & 0x000000ff;
		hash[i + 4]  = (ctx->state[1] >> (24 - i * 8)) & 0x000000ff;
		hash[i + 8]  = (ctx->state[2] >> (24 - i * 8)) & 0x000000ff;
		hash[i + 12] = (ctx->state[3] >> (24 - i * 8)) & 0x000000ff;
		hash[i + 16] = (ctx->state[4] >> (24 - i * 8)) & 0x000000ff;
		hash[i + 20] = (ctx->state[5] >> (24 - i * 8)) & 0x000000ff;
		hash[i + 24] = (ctx->state[6] >> (24 - i * 8)) & 0x000000ff;
		hash[i + 28] = (ctx->state[7] >> (24 - i * 8)) & 0x000000ff;
	}
}


__host__ void h_sha256_init(SHA256_CTX *ctx)
{
	ctx->datalen = 0;
	ctx->bitlen = 0;
	ctx->state[0] = 0x6a09e667;
	ctx->state[1] = 0xbb67ae85;
	ctx->state[2] = 0x3c6ef372;
	ctx->state[3] = 0xa54ff53a;
	ctx->state[4] = 0x510e527f;
	ctx->state[5] = 0x9b05688c;
	ctx->state[6] = 0x1f83d9ab;
	ctx->state[7] = 0x5be0cd19;
}

__host__ void h_sha256_update(SHA256_CTX *ctx, const char data[], size_t len)
{
	WORD i;

	for (i = 0; i < len; ++i) {
		ctx->data[ctx->datalen] = data[i];
		ctx->datalen++;
		if (ctx->datalen == 64) {
			h_sha256_transform(ctx, ctx->data);
			ctx->bitlen += 512;
			ctx->datalen = 0;
		}
	}
}

__host__ void h_sha256_final(SHA256_CTX *ctx, char hash[])
{
	WORD i;

	i = ctx->datalen;

	// Pad whatever data is left in the buffer.
	if (ctx->datalen < 56) {
		ctx->data[i++] = 0x80;
		while (i < 56)
			ctx->data[i++] = 0x00;
	}
	else {
		ctx->data[i++] = 0x80;
		while (i < 64)
			ctx->data[i++] = 0x00;
		h_sha256_transform(ctx, ctx->data);
		memset(ctx->data, 0, 56);
	}

	// Append to the padding the total message's length in bits and transform.
	ctx->bitlen += ctx->datalen * 8;
	ctx->data[63] = ctx->bitlen;
	ctx->data[62] = ctx->bitlen >> 8;
	ctx->data[61] = ctx->bitlen >> 16;
	ctx->data[60] = ctx->bitlen >> 24;
	ctx->data[59] = ctx->bitlen >> 32;
	ctx->data[58] = ctx->bitlen >> 40;
	ctx->data[57] = ctx->bitlen >> 48;
	ctx->data[56] = ctx->bitlen >> 56;
	h_sha256_transform(ctx, ctx->data);

	// Since this implementation uses little endian byte ordering and SHA uses big endian,
	// reverse all the bytes when copying the final state to the output hash.
	for (i = 0; i < 4; ++i) {
		hash[i]      = (ctx->state[0] >> (24 - i * 8)) & 0x000000ff;
		hash[i + 4]  = (ctx->state[1] >> (24 - i * 8)) & 0x000000ff;
		hash[i + 8]  = (ctx->state[2] >> (24 - i * 8)) & 0x000000ff;
		hash[i + 12] = (ctx->state[3] >> (24 - i * 8)) & 0x000000ff;
		hash[i + 16] = (ctx->state[4] >> (24 - i * 8)) & 0x000000ff;
		hash[i + 20] = (ctx->state[5] >> (24 - i * 8)) & 0x000000ff;
		hash[i + 24] = (ctx->state[6] >> (24 - i * 8)) & 0x000000ff;
		hash[i + 28] = (ctx->state[7] >> (24 - i * 8)) & 0x000000ff;
	}
}


__host__ __device__ void print_hash(unsigned char * buf)
{
   for (int i = 0; i < 32; i++){
      unsigned int hexnum = (unsigned int) buf[i];
      printf("%02x", hexnum);
   } 
   printf("\n");
   return;
}
__host__ __device__ int strLength(char * str)
{

   int count = 0;
   while (str[count] != '\n')
       count ++;
   printf("\nStrlen of %d", count);
   return count;
}

__host__ __device__ void strcpy(char * a, char * b)
{
   for (int i = 0; i < strLength(b); i++){
       a[i] = b[i];
       a[i + 1] = '\0';
   }


}
__device__ void sha256_hash(unsigned char * str)
{
	unsigned char text1[] = {"000000004c6fe27a1151135df1b1f5d36bc37b6455106e2fc64a8affb4518ddc"}; 
//	char text2[] = {"abcdbcdecdefdefgefghfghighijhijkijkljklmklmnlmnomnopnopq"};
//	char text3[] = {"aaaaaaaaaa"};
//	char hash1[SHA256_BLOCK_SIZE] = {0xba,0x78,0x16,0xbf,0x8f,0x01,0xcf,0xea,0x41,0x41,0x40,0xde,0x5d,0xae,0x22,0x23,
//	                                 0xb0,0x03,0x61,0xa3,0x96,0x17,0x7a,0x9c,0xb4,0x10,0xff,0x61,0xf2,0x00,0x15,0xad};
//	char hash2[SHA256_BLOCK_SIZE] = {0x24,0x8d,0x6a,0x61,0xd2,0x06,0x38,0xb8,0xe5,0xc0,0x26,0x93,0x0c,0x3e,0x60,0x39,
//	                                 0xa3,0x3c,0xe4,0x59,0x64,0xff,0x21,0x67,0xf6,0xec,0xed,0xd4,0x19,0xdb,0x06,0xc1};
//	char hash3[SHA256_BLOCK_SIZE] = {0xcd,0xc7,0x6e,0x5c,0x99,0x14,0xfb,0x92,0x81,0xa1,0xc7,0xe2,0x84,0xd7,0x3e,0x67,
//	                                 0xf1,0x80,0x9a,0x48,0xa4,0x97,0x20,0x0e,0x04,0x6d,0x39,0xcc,0xc7,0x11,0x2c,0xd0};
      	unsigned char buf[SHA256_BLOCK_SIZE];
	SHA256_CTX ctx;
//        str  = "00000000308971eee4b34bf76a3eda47bbfbdf1d0cf407a5ed6daf182f4b23b8"; 
	//int idx;
	//int pass = 1;
        
        printf("Starting sha hash with string -%s-\n", str);
        print_hash(str);
        printf("HERE");
        char hash_str[100];
        int n = 0;
        while (str[n] != '\0' || n < SHA256_BLOCK_SIZE)
        {
            hash_str[n] = (char) str[n];
            n ++;
        }
        str[n] = '\0';
        printf("Copied %d bytes\n", n);
        printf("Here is the hash_str\n");
        for (int i = 0; i < n; i ++)
            printf("%d-", hash_str[i]);
//        printf(hash_str);
//        printf("\n");
        //print_hash((unsigned char *)hash_str);
	//print_hash(str);
//        printf("\nHash length - %d", strLength((char*)hash_str));
        sha256_init(&ctx);
        printf("Finishing init");
	sha256_update(&ctx, (char*)text1, SHA256_BLOCK_SIZE );
        printf("Finishing update");
	sha256_final(&ctx, (char *) buf);
        printf("Finished hash");
        int difficulty = 33;
        bool invalid = false;
        //printf("%x", buf);
	for (int i = 0; i < 32; i ++){
//           printf("%c", (unsigned char*) buf[i]);
           unsigned int hexnum = (unsigned int) buf[i];
//           printf("%x-", (unsigned int) hexnum );
           for (int j = 128; j >= 1; j= j / 2){
//              printf("%d", hexnum & j);
              if (((int)hexnum & j) != 0){
           //      printf("1");
                 invalid = true;
              } else {
           //      printf("0");
                 difficulty --;
                 if (difficulty == 0)
                      break;
              }
              if (invalid || difficulty == 0)
                  break;
              //printf("%d", ((unsigned int) buf[i]) & j); 
           }
           if (invalid || difficulty == 0)
                break;
           //printf("%d", (unsigned char*) buf[i]);
           //printf("\nNext Bits\n");
        }
        printf("Printing hash \n");
        print_hash(buf);
        printf("Finished printing hash");
        if (invalid){
             printf("Not enough work done %d\n", difficulty);
             buf[0] = '\0';
            
             memcpy(str, buf, SHA256_BLOCK_SIZE);
        }else{
             printf("YAY you found one");
             print_hash(buf);
             memcpy(str, buf, SHA256_BLOCK_SIZE);
        }
}


__host__ void h_sha256_hash(char * str)
{
      	unsigned char buf[SHA256_BLOCK_SIZE];
	SHA256_CTX ctx;
        printf("Starting sha hash \n");
	h_sha256_init(&ctx);
        printf("Finishing init");
	h_sha256_update(&ctx, str, strLength(str));
        printf("Finishing update");
	h_sha256_final(&ctx, (char *) buf);
        printf("Finished hash");
        int difficulty = 33;
        bool invalid = false;
	for (int i = 0; i < 32; i ++){
           unsigned int hexnum = (unsigned int) buf[i];
           for (int j = 128; j >= 1; j= j / 2){
              if (((int)hexnum & j) != 0){
                 invalid = true;
              } else {
                 difficulty --;
                 if (difficulty == 0)
                      break;
              }
              if (invalid || difficulty == 0)
                  break;
              //printf("%d", ((unsigned int) buf[i]) & j); 
           }
           if (invalid || difficulty == 0)
                break;
           //printf("%d", (unsigned char*) buf[i]);
           //printf("\nNext Bits\n");
        }
        printf("Printing hash \n");
        print_hash(buf);
        printf("Finished printing hash");
        if (invalid){
             printf("Not enough work done %d\n", difficulty);
             buf[0] = '\0';
            
             memcpy(str, buf, SHA256_BLOCK_SIZE);
        }else{
             printf("YAY you found one");
             print_hash(buf);
             memcpy(str, buf, SHA256_BLOCK_SIZE);
        }
}


__global__ void testKernel(unsigned char *var)
{   
    printf("Yay in gpu mode\n");
    print_hash((unsigned char*)var);
//    printf("[%d, %d]:\t\tValue is:%s\n",\
//            blockIdx.y*gridDim.x+blockIdx.x,\
//            threadIdx.z*blockDim.x*blockDim.y+threadIdx.y*blockDim.x+threadIdx.x,\
//            (char*)var);
    sha256_hash(var);
    printf("End kernel");
}

/* 
 * error - wrapper for perror
 */
void error(char *msg) {
    perror(msg);
    exit(0);
}

void getTip(char * buf){
    int sockfd, portno, n;
    struct sockaddr_in serveraddr;
    struct hostent *server;
    char *hostname;
    //char buf[BUFSIZE];

    /* check command line arguments */
//    if (argc != 3) {
//       fprintf(stderr,"usage: %s <hostname> <port>\n", argv[0]);
      // exit(0);
//    }
//    hostname = argv[1];
    hostname = (char*) "localhost\0";
    hostname = (char*) "hubris.media.mit.edu\0";
//    portno = atoi(argv[2]);
    portno = 6262;

    /* socket: create the socket */
    sockfd = socket(AF_INET, SOCK_STREAM, 0);
    if (sockfd < 0)
        error((char *) "ERROR opening socket");

    /* gethostbyname: get the server's DNS entry */
    server = gethostbyname(hostname);
    if (server == NULL) {
        fprintf(stderr,"ERROR, no such host as %s\n", hostname);
        exit(0);
    }

    /* build the server's Internet address */
    bzero((char *) &serveraddr, sizeof(serveraddr));
    serveraddr.sin_family = AF_INET;
    bcopy((char *)server->h_addr,
          (char *)&serveraddr.sin_addr.s_addr, server->h_length);
    serveraddr.sin_port = htons(portno);

    /* connect: create a connection with the server */
    if (connect(sockfd, (struct sockaddr *) &serveraddr, sizeof(serveraddr)) < 0)
      error((char *) "ERROR connecting");

    /* get message line from the user */
//    printf("Please enter msg: ");
    bzero(buf, BUFSIZE);
    //fgets(buf, BUFSIZE, stdin);
    sprintf(buf, "TRQ\n");
    /* send the message line to the server */
    n = write(sockfd, buf, strlen(buf));
    if (n < 0)
      error((char *) "ERROR writing to socket");

    /* print the server's reply */
    bzero(buf, BUFSIZE);
    n = read(sockfd, buf, BUFSIZE);
    if (n < 0)
      error((char *) "ERROR reading from socket");
    printf("Returned tip: %s-----------", buf);
    close(sockfd);
    return;

}

int main(int argc, char **argv)
{
    int devID;
    hipDeviceProp_t props;

    // This will pick the best possible CUDA capable device
    devID = findCudaDevice(argc, (const char **)argv);

    //Get GPU information
    checkCudaErrors(hipGetDevice(&devID));
    checkCudaErrors(hipGetDeviceProperties(&props, devID));
    printf("Device %d: \"%s\" with Compute %d.%d capability\n",
           devID, props.name, props.major, props.minor);

    printf("printf() is called. Output:\n\n");

//    printf("SHA-256 tests: %s\n", sha256_test() ? "SUCCEEDED" : "FAILED");


    //Kernel configuration, where a two-dimensional grid and
    //three-dimensional blocks are configured.
//    dim3 dimGrid(2, 2);
//    dim3 dimBlock(2, 2, 2);
//    testKernel<<<dimGrid, dimBlock>>>(10);
    char *tip = (char*) malloc(BUFSIZE);
    unsigned char * h_tip = (unsigned char*) malloc(SHA256_BLOCK_SIZE);
    printf("Gettin tip");
    getTip(tip);
    for (int i=0; i < 100; i++)
       printf("%d_", tip[i]);
    printf("\n");

    h_sha256_hash(tip);
    for (int i= 0; i < SHA256_BLOCK_SIZE; i++)
        printf("%d_", tip[i]);
    printf("\n");
//    h_tip = (unsigned char *) tip;
    memcpy(h_tip, tip, SHA256_BLOCK_SIZE);
    print_hash((unsigned char*)tip);
    print_hash(h_tip);
    unsigned char* d_tip = NULL;
    hipError_t err = hipSuccess;    
    err = hipMalloc((void **)&d_tip, SHA256_BLOCK_SIZE);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }    

    err = hipMemcpy(d_tip, h_tip, SHA256_BLOCK_SIZE, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }


    testKernel<<<1,1>>>(d_tip);
    hipDeviceSynchronize();
    return EXIT_SUCCESS;
}

