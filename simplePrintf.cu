#include "hip/hip_runtime.h"
/*-256 tests: %s\n", sha256_test() ? "SUCCEEDED" : "FAILED");

 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */


// System includes
#include <stdio.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// helper functions and utilities to work with CUDA
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
#include <timer.h>

// TCP
#include <stdlib.h>
#include <string.h>
#include <unistd.h>
#include <sys/types.h>
#include <sys/socket.h>
#include <netinet/in.h>
#include <netdb.h>

#define BUFSIZE 1024

#ifndef MAX
#define MAX(a,b) (a > b ? a : b)
#endif


typedef unsigned int  WORD;             // 32-bit word, change to "long" for 16-bit machines

typedef struct {
	unsigned char data[64];
	WORD datalen;
	unsigned long long bitlen;
	WORD state[8];
} SHA256_CTX;

#define SHA256_BLOCK_SIZE 32            // SHA256 outputs a 32 byte digest

/*********************************************************************
* Filename:   sha256.c
* Author:     Brad Conte (brad AT bradconte.com)
* Copyright:
* Disclaimer: This code is presented "as is" without any guarantees.
* Details:    Implementation of the SHA-256 hashing algorithm.
              SHA-256 is one of the three algorithms in the SHA2
              specification. The others, SHA-384 and SHA-512, are not
              offered in this implementation.
              Algorithm specification can be found here:
               * http://csrc.nist.gov/publications/fips/fips180-2/fips180-2withchangenotice.pdf
              This implementation uses little endian byte order.
*********************************************************************/

/*************************** HEADER FILES ***************************/
#include <memory.h>

/****************************** MACROS ******************************/
#define ROTLEFT(a,b) (((a) << (b)) | ((a) >> (32-(b))))
#define ROTRIGHT(a,b) (((a) >> (b)) | ((a) << (32-(b))))

#define CH(x,y,z) (((x) & (y)) ^ (~(x) & (z)))
#define MAJ(x,y,z) (((x) & (y)) ^ ((x) & (z)) ^ ((y) & (z)))
#define EP0(x) (ROTRIGHT(x,2) ^ ROTRIGHT(x,13) ^ ROTRIGHT(x,22))
#define EP1(x) (ROTRIGHT(x,6) ^ ROTRIGHT(x,11) ^ ROTRIGHT(x,25))
#define SIG0(x) (ROTRIGHT(x,7) ^ ROTRIGHT(x,18) ^ ((x) >> 3))
#define SIG1(x) (ROTRIGHT(x,17) ^ ROTRIGHT(x,19) ^ ((x) >> 10))

/**************************** VARIABLES *****************************/
__device__ static const WORD k[64] = {
	0x428a2f98,0x71374491,0xb5c0fbcf,0xe9b5dba5,0x3956c25b,0x59f111f1,0x923f82a4,0xab1c5ed5,
	0xd807aa98,0x12835b01,0x243185be,0x550c7dc3,0x72be5d74,0x80deb1fe,0x9bdc06a7,0xc19bf174,
	0xe49b69c1,0xefbe4786,0x0fc19dc6,0x240ca1cc,0x2de92c6f,0x4a7484aa,0x5cb0a9dc,0x76f988da,
	0x983e5152,0xa831c66d,0xb00327c8,0xbf597fc7,0xc6e00bf3,0xd5a79147,0x06ca6351,0x14292967,
	0x27b70a85,0x2e1b2138,0x4d2c6dfc,0x53380d13,0x650a7354,0x766a0abb,0x81c2c92e,0x92722c85,
	0xa2bfe8a1,0xa81a664b,0xc24b8b70,0xc76c51a3,0xd192e819,0xd6990624,0xf40e3585,0x106aa070,
	0x19a4c116,0x1e376c08,0x2748774c,0x34b0bcb5,0x391c0cb3,0x4ed8aa4a,0x5b9cca4f,0x682e6ff3,
	0x748f82ee,0x78a5636f,0x84c87814,0x8cc70208,0x90befffa,0xa4506ceb,0xbef9a3f7,0xc67178f2
};

static const WORD h_k[64] = {
	0x428a2f98,0x71374491,0xb5c0fbcf,0xe9b5dba5,0x3956c25b,0x59f111f1,0x923f82a4,0xab1c5ed5,
	0xd807aa98,0x12835b01,0x243185be,0x550c7dc3,0x72be5d74,0x80deb1fe,0x9bdc06a7,0xc19bf174,
	0xe49b69c1,0xefbe4786,0x0fc19dc6,0x240ca1cc,0x2de92c6f,0x4a7484aa,0x5cb0a9dc,0x76f988da,
	0x983e5152,0xa831c66d,0xb00327c8,0xbf597fc7,0xc6e00bf3,0xd5a79147,0x06ca6351,0x14292967,
	0x27b70a85,0x2e1b2138,0x4d2c6dfc,0x53380d13,0x650a7354,0x766a0abb,0x81c2c92e,0x92722c85,
	0xa2bfe8a1,0xa81a664b,0xc24b8b70,0xc76c51a3,0xd192e819,0xd6990624,0xf40e3585,0x106aa070,
	0x19a4c116,0x1e376c08,0x2748774c,0x34b0bcb5,0x391c0cb3,0x4ed8aa4a,0x5b9cca4f,0x682e6ff3,
	0x748f82ee,0x78a5636f,0x84c87814,0x8cc70208,0x90befffa,0xa4506ceb,0xbef9a3f7,0xc67178f2
};

/*********************** FUNCTION DEFINITIONS ***********************/
 __device__ void sha256_transform(SHA256_CTX *ctx, const unsigned char data[])
{
	WORD a, b, c, d, e, f, g, h, i, j, t1, t2, m[64];

	for (i = 0, j = 0; i < 16; ++i, j += 4)
		m[i] = (data[j] << 24) | (data[j + 1] << 16) | (data[j + 2] << 8) | (data[j + 3]);
	for ( ; i < 64; ++i)
		m[i] = SIG1(m[i - 2]) + m[i - 7] + SIG0(m[i - 15]) + m[i - 16];

	a = ctx->state[0];
	b = ctx->state[1];
	c = ctx->state[2];
	d = ctx->state[3];
	e = ctx->state[4];
	f = ctx->state[5];
	g = ctx->state[6];
	h = ctx->state[7];
	for (i = 0; i < 64; ++i) {
                t1 = h + EP1(e) + CH(e,f,g) + k[i] + m[i];
		t2 = EP0(a) + MAJ(a,b,c);
		h = g;
		g = f;
		f = e;
		e = d + t1;
		d = c;
		c = b;
		b = a;
		a = t1 + t2;
	}
	ctx->state[0] += a;
	ctx->state[1] += b;
	ctx->state[2] += c;
	ctx->state[3] += d;
	ctx->state[4] += e;
	ctx->state[5] += f;
	ctx->state[6] += g;
	ctx->state[7] += h;
}

 __host__ void h_sha256_transform(SHA256_CTX *ctx, const unsigned char data[])
{
	WORD a, b, c, d, e, f, g, h, i, j, t1, t2, m[64];

	for (i = 0, j = 0; i < 16; ++i, j += 4)
		m[i] = (data[j] << 24) | (data[j + 1] << 16) | (data[j + 2] << 8) | (data[j + 3]);
	for ( ; i < 64; ++i)
		m[i] = SIG1(m[i - 2]) + m[i - 7] + SIG0(m[i - 15]) + m[i - 16];

	a = ctx->state[0];
	b = ctx->state[1];
	c = ctx->state[2];
	d = ctx->state[3];
	e = ctx->state[4];
	f = ctx->state[5];
	g = ctx->state[6];
	h = ctx->state[7];
	for (i = 0; i < 64; ++i) {
                t1 = h + EP1(e) + CH(e,f,g) + h_k[i] + m[i];
		t2 = EP0(a) + MAJ(a,b,c);
		h = g;
		g = f;
		f = e;
		e = d + t1;
		d = c;
		c = b;
		b = a;
		a = t1 + t2;
	}
	ctx->state[0] += a;
	ctx->state[1] += b;
	ctx->state[2] += c;
	ctx->state[3] += d;
	ctx->state[4] += e;
	ctx->state[5] += f;
	ctx->state[6] += g;
	ctx->state[7] += h;
}
__device__ void sha256_init(SHA256_CTX *ctx)
{
	ctx->datalen = 0;
	ctx->bitlen = 0;
	ctx->state[0] = 0x6a09e667;
	ctx->state[1] = 0xbb67ae85;
	ctx->state[2] = 0x3c6ef372;
	ctx->state[3] = 0xa54ff53a;
	ctx->state[4] = 0x510e527f;
	ctx->state[5] = 0x9b05688c;
	ctx->state[6] = 0x1f83d9ab;
	ctx->state[7] = 0x5be0cd19;
}

__device__ void sha256_update(SHA256_CTX *ctx, const char data[], size_t len)
{
	WORD i;

	for (i = 0; i < len; ++i) {
		ctx->data[ctx->datalen] = data[i];
		ctx->datalen++;
		if (ctx->datalen == 64) {
			sha256_transform(ctx, ctx->data);
			ctx->bitlen += 512;
			ctx->datalen = 0;
		}
	}
}

__device__ void sha256_final(SHA256_CTX *ctx, char hash[])
{
	WORD i;

	i = ctx->datalen;

	// Pad whatever data is left in the buffer.
	if (ctx->datalen < 56) {
		ctx->data[i++] = 0x80;
		while (i < 56)
			ctx->data[i++] = 0x00;
	}
	else {
		ctx->data[i++] = 0x80;
		while (i < 64)
			ctx->data[i++] = 0x00;
		sha256_transform(ctx, ctx->data);
		memset(ctx->data, 0, 56);
	}

	// Append to the padding the total message's length in bits and transform.
	ctx->bitlen += ctx->datalen * 8;
	ctx->data[63] = ctx->bitlen;
	ctx->data[62] = ctx->bitlen >> 8;
	ctx->data[61] = ctx->bitlen >> 16;
	ctx->data[60] = ctx->bitlen >> 24;
	ctx->data[59] = ctx->bitlen >> 32;
	ctx->data[58] = ctx->bitlen >> 40;
	ctx->data[57] = ctx->bitlen >> 48;
	ctx->data[56] = ctx->bitlen >> 56;
	sha256_transform(ctx, ctx->data);

	// Since this implementation uses little endian byte ordering and SHA uses big endian,
	// reverse all the bytes when copying the final state to the output hash.
	for (i = 0; i < 4; ++i) {
		hash[i]      = (ctx->state[0] >> (24 - i * 8)) & 0x000000ff;
		hash[i + 4]  = (ctx->state[1] >> (24 - i * 8)) & 0x000000ff;
		hash[i + 8]  = (ctx->state[2] >> (24 - i * 8)) & 0x000000ff;
		hash[i + 12] = (ctx->state[3] >> (24 - i * 8)) & 0x000000ff;
		hash[i + 16] = (ctx->state[4] >> (24 - i * 8)) & 0x000000ff;
		hash[i + 20] = (ctx->state[5] >> (24 - i * 8)) & 0x000000ff;
		hash[i + 24] = (ctx->state[6] >> (24 - i * 8)) & 0x000000ff;
		hash[i + 28] = (ctx->state[7] >> (24 - i * 8)) & 0x000000ff;
	}
}


__host__ void h_sha256_init(SHA256_CTX *ctx)
{
	ctx->datalen = 0;
	ctx->bitlen = 0;
	ctx->state[0] = 0x6a09e667;
	ctx->state[1] = 0xbb67ae85;
	ctx->state[2] = 0x3c6ef372;
	ctx->state[3] = 0xa54ff53a;
	ctx->state[4] = 0x510e527f;
	ctx->state[5] = 0x9b05688c;
	ctx->state[6] = 0x1f83d9ab;
	ctx->state[7] = 0x5be0cd19;
}

__host__ void h_sha256_update(SHA256_CTX *ctx, const char data[], size_t len)
{
	WORD i;

	for (i = 0; i < len; ++i) {
		ctx->data[ctx->datalen] = data[i];
		ctx->datalen++;
		if (ctx->datalen == 64) {
			h_sha256_transform(ctx, ctx->data);
			ctx->bitlen += 512;
			ctx->datalen = 0;
		}
	}
}

__host__ void h_sha256_final(SHA256_CTX *ctx, char hash[])
{
	WORD i;

	i = ctx->datalen;

	// Pad whatever data is left in the buffer.
	if (ctx->datalen < 56) {
		ctx->data[i++] = 0x80;
		while (i < 56)
			ctx->data[i++] = 0x00;
	}
	else {
		ctx->data[i++] = 0x80;
		while (i < 64)
			ctx->data[i++] = 0x00;
		h_sha256_transform(ctx, ctx->data);
		memset(ctx->data, 0, 56);
	}

	// Append to the padding the total message's length in bits and transform.
	ctx->bitlen += ctx->datalen * 8;
	ctx->data[63] = ctx->bitlen;
	ctx->data[62] = ctx->bitlen >> 8;
	ctx->data[61] = ctx->bitlen >> 16;
	ctx->data[60] = ctx->bitlen >> 24;
	ctx->data[59] = ctx->bitlen >> 32;
	ctx->data[58] = ctx->bitlen >> 40;
	ctx->data[57] = ctx->bitlen >> 48;
	ctx->data[56] = ctx->bitlen >> 56;
	h_sha256_transform(ctx, ctx->data);

	// Since this implementation uses little endian byte ordering and SHA uses big endian,
	// reverse all the bytes when copying the final state to the output hash.
	for (i = 0; i < 4; ++i) {
		hash[i]      = (ctx->state[0] >> (24 - i * 8)) & 0x000000ff;
		hash[i + 4]  = (ctx->state[1] >> (24 - i * 8)) & 0x000000ff;
		hash[i + 8]  = (ctx->state[2] >> (24 - i * 8)) & 0x000000ff;
		hash[i + 12] = (ctx->state[3] >> (24 - i * 8)) & 0x000000ff;
		hash[i + 16] = (ctx->state[4] >> (24 - i * 8)) & 0x000000ff;
		hash[i + 20] = (ctx->state[5] >> (24 - i * 8)) & 0x000000ff;
		hash[i + 24] = (ctx->state[6] >> (24 - i * 8)) & 0x000000ff;
		hash[i + 28] = (ctx->state[7] >> (24 - i * 8)) & 0x000000ff;
	}
}


__host__ __device__ void print_hash(unsigned char * buf)
{
   for (int i = 0; i < 32; i++){
      unsigned int hexnum = (unsigned int) buf[i];
      printf("%02x", hexnum);
   } 
   printf("\n");
   return;
}
__host__ int strlength(char * str)
{

   int count = 0;
   while (str[count] != '\n')
       count ++;
   return count;
}


__device__ int strlengthzero(char * str)
{
   int count = 0;
   while (str[count] != '\0')
       count ++;
   return count;
}

__device__ void get_ending(unsigned char * buf, int threadID, int offset, int d_num)
{
   int n = 0;
   char name[9] = {" turtle "};
   bool first = true;
   for (int i = 0; i < 8; i++){
       buf[n] = name[i];
       n++;
   }
   buf[n] = '0' + d_num;
   n++;
   buf[n] = '/';
   n++;
   while (threadID != 0 || first){
       first = false;
       int nextNum = threadID % 10;
       unsigned char nextChar = '0' + nextNum;
       buf[n] = nextChar;
       n++;
       threadID = threadID / 10;
   } 
   buf[n] = '/';
   n++;
   first = true;
   while (offset != 0 || first) {
      first = false;
      int nextNum = offset % 10;
      unsigned char nextChar = '0' + nextNum;
      buf[n] = nextChar;
      n++;
      offset = offset / 10;
   }
   buf[n] = '\0';
}

__device__ void sha256_hash(unsigned char * str, unsigned char * result, int threadNum, int offset, int d_num)
{
      	unsigned char buf[SHA256_BLOCK_SIZE];
	SHA256_CTX ctx;

        //printf("Starting sha hash with string -%s-\n", str);
        //print_hash(str);
        char hash_str[100];
        int n_len = 64;
        for (int n = 0; n < 32; n++)
        {
            unsigned int fullBits = (unsigned int) str[n];
            unsigned int leftBit = (fullBits >> 4);
            unsigned int rightBit = (fullBits & 0xF);
            if (leftBit < 10)
                hash_str[n * 2] = '0' + leftBit;
            else
                hash_str[n * 2] = 'a' - 10 + leftBit;
            if (rightBit < 10)
                hash_str[n * 2 + 1] = '0' + rightBit;
            else
                hash_str[n * 2 + 1] = 'a' - 10 + rightBit;
        }
        
        char ending[36];
        get_ending((unsigned char*) ending, threadNum, offset, d_num);
        for (int i = 0; i < strlengthzero(ending); i++){
            hash_str[64 + i] = ending[i];
            n_len ++;
        }
//        printf("Here is the hash_str: %s\n", hash_str);
        sha256_init(&ctx);
	sha256_update(&ctx, hash_str, n_len);
	sha256_final(&ctx, (char *) buf);
        int difficulty = 33;
        //int difficulty = 20;
        bool invalid = false;
	for (int i = 0; i < 32; i ++){
           unsigned int hexnum = (unsigned int) buf[i];
           for (int j = 128; j >= 1; j= j / 2){
              if (((int)hexnum & j) != 0){
                 invalid = true;
              } else {
                 difficulty --;
                 if (difficulty == 0)
                      break;
              }
              if (invalid || difficulty == 0)
                  break;
           }
           if (invalid || difficulty == 0)
                break;
        }
        if (offset % 100 == 0 && threadNum == 0)
            printf("%s\n",hash_str);
        if (invalid){
             //printf("Not enough work done %d\n", difficulty);
             buf[0] = '\0';
        }else{
             printf("YAY you found one: %s\n", hash_str);

             memcpy(result, hash_str, n_len);
        }
}


__host__ void h_sha256_hash(char * str)
{
      	unsigned char buf[SHA256_BLOCK_SIZE];
	SHA256_CTX ctx;
	h_sha256_init(&ctx);
	h_sha256_update(&ctx, str, strlength(str));
	h_sha256_final(&ctx, (char *) buf);
        int difficulty = 33;
        bool invalid = false;
	for (int i = 0; i < 32; i ++){
           unsigned int hexnum = (unsigned int) buf[i];
           for (int j = 128; j >= 1; j= j / 2){
              if (((int)hexnum & j) != 0){
                 invalid = true;
              } else {
                 difficulty --;
                 if (difficulty == 0)
                      break;
              }
              if (invalid || difficulty == 0)
                  break;
           }
           if (invalid || difficulty == 0)
                break;
        }
        if (invalid){
             buf[0] = '\0';
             memcpy(str, buf, SHA256_BLOCK_SIZE);
        }else{
             //printf("enough work done to satisfy difficulty \n");
             memcpy(str, buf, SHA256_BLOCK_SIZE);
        }
}


__global__ void testKernel(unsigned char *var, unsigned char * result, int offset, int d_num)
{   
    bool first = true; 
    while (offset % 100 != 0 || first)
    {
        first = false;
        int threadNum = blockDim.x * blockIdx.x + threadIdx.x;
        //printf("Yay in gpu mode, Thread: %d\n", threadNum);
        sha256_hash(var, result, threadNum,  offset, d_num);
        offset ++;
        if (result[0] != '\0')
            break;
    }
}

/* 
 * error - wrapper for perror
 */
void error(char *msg) {
    perror(msg);
    exit(0);
}

void sendBlock(char * block)
{

    int sockfd, portno, n;
    struct sockaddr_in serveraddr;
    struct hostent *server;
    char *hostname;
    char buf[BUFSIZE];

    /* check command line arguments */
//    if (argc != 3) {
//       fprintf(stderr,"usage: %s <hostname> <port>\n", argv[0]);
      // exit(0);
//    }
//    hostname = argv[1];
    hostname = (char*) "localhost\0";
    hostname = (char*) "hubris.media.mit.edu\0";
//    portno = atoi(argv[2]);
    portno = 6262;

    /* socket: create the socket */
    sockfd = socket(AF_INET, SOCK_STREAM, 0);
    if (sockfd < 0)
        error((char *) "ERROR opening socket");
    printf("Opened socket\n");
    /* gethostbyname: get the server's DNS entry */
    server = gethostbyname(hostname);
    if (server == NULL) {
        fprintf(stderr,"ERROR, no such host as %s\n", hostname);
        exit(0);
    }

    /* build the server's Internet address */
    bzero((char *) &serveraddr, sizeof(serveraddr));
    serveraddr.sin_family = AF_INET;
    bcopy((char *)server->h_addr,
          (char *)&serveraddr.sin_addr.s_addr, server->h_length);
    serveraddr.sin_port = htons(portno);

    /* connect: create a connection with the server */
    if (connect(sockfd, (struct sockaddr *) &serveraddr, sizeof(serveraddr)) < 0)
      error((char *) "ERROR connecting");

    /* get message line from the user */
//    printf("Please enter msg: ");
    //fgets(buf, BUFSIZE, stdin);
    /* send the message line to the server */
    bzero(buf, BUFSIZE);  
    sprintf(buf, "%s\n", block);
    printf("Wrote result to buf\n");
    n = write(sockfd, buf, strlen(buf));
    if (n < 0)
      error((char *) "ERROR writing to socket");

    /* print the server's reply */
    bzero(buf, BUFSIZE);
    n = read(sockfd, buf, BUFSIZE);
    if (n < 0)
      error((char *) "ERROR reading from socket");
    printf("Returned tip: %s-----------", buf);
    close(sockfd);
    return;




}

int getScore()
{
    int sockfd, portno, n;
    struct sockaddr_in serveraddr;
    struct hostent *server;
    char *hostname;
    char buf[BUFSIZE];

    /* check command line arguments */
//    if (argc != 3) {
//       fprintf(stderr,"usage: %s <hostname> <port>\n", argv[0]);
      // exit(0);
//    }
//    hostname = argv[1];
    hostname = (char*) "localhost\0";
    hostname = (char*) "hubris.media.mit.edu\0";
//    portno = atoi(argv[2]);
    portno = 6299;

    /* socket: create the socket */
    sockfd = socket(AF_INET, SOCK_STREAM, 0);
    if (sockfd < 0)
        error((char *) "ERROR opening socket");

    /* gethostbyname: get the server's DNS entry */
    server = gethostbyname(hostname);
    if (server == NULL) {
        fprintf(stderr,"ERROR, no such host as %s\n", hostname);
        exit(0);
    }

    /* build the server's Internet address */
    bzero((char *) &serveraddr, sizeof(serveraddr));
    serveraddr.sin_family = AF_INET;
    bcopy((char *)server->h_addr,
          (char *)&serveraddr.sin_addr.s_addr, server->h_length);
    serveraddr.sin_port = htons(portno);

    /* connect: create a connection with the server */
    if (connect(sockfd, (struct sockaddr *) &serveraddr, sizeof(serveraddr)) < 0)
      error((char *) "ERROR connecting");

    /* get message line from the user */
//    printf("Please enter msg: ");
    bzero(buf, BUFSIZE);
    //fgets(buf, BUFSIZE, stdin);
    sprintf(buf, "\n");
    /* send the message line to the server */
    n = write(sockfd, buf, strlen(buf));
    if (n < 0)
      error((char *) "ERROR writing to socket");

    /* print the server's reply */
    bzero(buf, BUFSIZE);
    n = read(sockfd, buf, BUFSIZE);
    if (n < 0)
      error((char *) "ERROR reading from socket");
    //printf("Returned tip: %s", buf);
    printf("Current Score: ");
    for (int i = 51; i < 55; i++)
        printf("%c", buf[i]);
    printf("\n");
    close(sockfd);
    if (buf[51] == '1' && buf[52] == '3' && buf[53] == '3' && buf[54] == '7')
        return 1;

    return 0;

}

void getTip(char * buf){
    int sockfd, portno, n;
    struct sockaddr_in serveraddr;
    struct hostent *server;
    char *hostname;
    //char buf[BUFSIZE];

    /* check command line arguments */
//    if (argc != 3) {
//       fprintf(stderr,"usage: %s <hostname> <port>\n", argv[0]);
      // exit(0);
//    }
//    hostname = argv[1];
    hostname = (char*) "localhost\0";
    hostname = (char*) "hubris.media.mit.edu\0";
//    portno = atoi(argv[2]);
    portno = 6262;

    /* socket: create the socket */
    sockfd = socket(AF_INET, SOCK_STREAM, 0);
    if (sockfd < 0)
        error((char *) "ERROR opening socket");

    /* gethostbyname: get the server's DNS entry */
    server = gethostbyname(hostname);
    if (server == NULL) {
        fprintf(stderr,"ERROR, no such host as %s\n", hostname);
        exit(0);
    }

    /* build the server's Internet address */
    bzero((char *) &serveraddr, sizeof(serveraddr));
    serveraddr.sin_family = AF_INET;
    bcopy((char *)server->h_addr,
          (char *)&serveraddr.sin_addr.s_addr, server->h_length);
    serveraddr.sin_port = htons(portno);

    /* connect: create a connection with the server */
    if (connect(sockfd, (struct sockaddr *) &serveraddr, sizeof(serveraddr)) < 0)
      error((char *) "ERROR connecting");

    /* get message line from the user */
//    printf("Please enter msg: ");
    bzero(buf, BUFSIZE);
    //fgets(buf, BUFSIZE, stdin);
    sprintf(buf, "TRQ\n");
    /* send the message line to the server */
    n = write(sockfd, buf, strlen(buf));
    if (n < 0)
      error((char *) "ERROR writing to socket");

    /* print the server's reply */
    bzero(buf, BUFSIZE);
    n = read(sockfd, buf, BUFSIZE);
    if (n < 0)
      error((char *) "ERROR reading from socket");
    printf("Returned tip: %s", buf);
    close(sockfd);
    return;
}

int main(int argc, char **argv)
{
    int GPU_N;
    checkCudaErrors(hipGetDeviceCount(&GPU_N));
    printf("CUDA-capable device count: %i\n", GPU_N);
    if(argc < 2)
    {
        printf("Missing argument, use -device to set gpu slot\n");
        exit(-1);
    }

    int devID;
    hipDeviceProp_t props;

    // This will pick the best possible CUDA capable device
    devID = findCudaDevice(argc, (const char **)argv);
    printf("Device: %d\n", devID);
    //Get GPU information
    checkCudaErrors(hipGetDevice(&devID));
    checkCudaErrors(hipGetDeviceProperties(&props, devID));
    printf("Device %d: \"%s\" with Compute %d.%d capability\n",
           devID, props.name, props.major, props.minor);
    int counter = 0;

    int offset = 0;
    char *lastTip = (char*) malloc(BUFSIZE);
    bzero(lastTip, BUFSIZE);
  while(true){
    counter ++;
    if (counter % 5 == 0){
        if (getScore())
            exit(0);
    }
    char *tip = (char*) malloc(BUFSIZE);
    unsigned char * h_tip = (unsigned char*) malloc(SHA256_BLOCK_SIZE);
    getTip(tip);
    if (tip[14] != lastTip[14])
    {
        printf("Last tip - %s\n", lastTip);
        printf("New tip  - %s\n", tip);
        offset = 0;
        printf("Got new Tip \n ");
    }
    strcpy(lastTip, tip);
    h_sha256_hash(tip);

    memcpy(h_tip, tip, SHA256_BLOCK_SIZE);
    unsigned char* d_tip = NULL;
    hipError_t err = hipSuccess;    
    err = hipMalloc((void **)&d_tip, SHA256_BLOCK_SIZE);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }    

    err = hipMemcpy(d_tip, h_tip, SHA256_BLOCK_SIZE, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    unsigned char * result = NULL;
    unsigned char *h_result = (unsigned char*) malloc(100);
    bzero(h_result, 100);
    err = hipMalloc((void **)&result, 100);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }    
    
    err = hipMemcpy(result, h_result, 100, hipMemcpyHostToDevice);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    StartTimer();
    testKernel<<<1024,1024>>>(d_tip, result, offset, devID);
    offset += 100;
    hipDeviceSynchronize();
    printf("  GPU Processing time: %f (ms)\n\n", GetTimer());
    
    bzero(h_result, 100);
    err = hipMemcpy(h_result, result, 100, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    if (h_result[0] != '\0')
    {
        printf("result detected: %s\n", h_result);
        sendBlock((char*) h_result);
    }
  }
}

